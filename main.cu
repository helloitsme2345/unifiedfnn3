#include "hip/hip_runtime.h"


/* Include Files */
#include "main.h"
#include "InformationFilterUpdate.h"
#include "stdio.h"
#include <time.h>

static void main_InformationFilterUpdate(void);



static void main_InformationFilterUpdate(void)
{
    double y_meas[13] = { 0.2018,-0.3185,-0.02881,7.27756,7.6464,7.4544,7.2981,7.214626736,0,0,7.550,7.2564,0 };
    double B_usedMeas_vec[13] = { 1,1,1,1,0,0,0,0,0,0,1,1,0 };
    double initialization_vec[4] = { 75.1,83.4,7.25,0.413 };
    // double Pk_init[64] = { 0.0100,0, 0, 0, 0, 0, 0, 0, 0, 0.0300, 0, 0,  0, 0, 0, 0, 0, 0, 0.0001, 0, 0, 0, 0, 0, 0, 0, 0, 0.0020, 0, 0, 0, 0, 0, 0 ,0 ,0 ,0.0100 ,0 ,0 ,0 ,0, 0, 0, 0,  0 ,0.0300 ,0 ,0, 0 ,0 ,0 ,0 ,0 ,0 ,0.0001 ,0, 0 ,0, 0, 0, 0 ,0 ,0 ,0.0020 };
    double delta = 0.000734;
    double Rw[8] = { 10,1,10,1, 10,1,10,1 };
    double Re[13] = { 1,1,0.001,1,1,1,1,1,1,1,1,1,1 };
    double L_imuToRear = 0.1;
    double L_geometricWheelbase = 3.7;
    double L_trackWidth[5] = { 2.05,0,1.85,0,0 };
    double L_axlePos[5] = { 0,0,-3.7,0,0 };
    double T = 0.01;
    double xk_m_out[16];
    double op[16];


    clock_t start = clock();
    InformationFilterUpdate(y_meas, B_usedMeas_vec, initialization_vec, delta,
        Rw, Re, L_imuToRear, L_geometricWheelbase, L_trackWidth,
        L_axlePos, T, xk_m_out, op);
    clock_t end = clock();
    double cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

    printf("for loop took %f seconds to execute \n", cpu_time_used);
    for (int i = 0; i < 8; i++)
    {
        printf("%lf ", xk_m_out[i]);
    }
    printf("\n");
    for (int i = 0; i < 16; i++)
    {
        printf("%lf ", op[i]);
    }
    printf("\n");

}


int main(int argc, const char* const argv[])
{
    (void)argc;
    (void)argv;


    main_InformationFilterUpdate();


    InformationFilterUpdate_terminate();


}