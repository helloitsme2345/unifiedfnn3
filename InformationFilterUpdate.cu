#include "hip/hip_runtime.h"
//
// File: InformationFilterUpdate.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 07-Sep-2020 10:58:04
//

// Include Files
#include "InformationFilterUpdate.h"
#include "MWCudaDimUtility.hpp"
#include "hipsolver.h"
#include <cmath>

// Variable Definitions
static unsigned int method;
static unsigned int state;
static unsigned int b_state[2];
static unsigned int c_state[625];
static boolean_T state_not_empty;
static boolean_T isInitialized_InformationFilterUpdate = false;

// Function Declarations
static __global__ void InformationFilterUpdate_kernel1(double Pk_init[256]);
static __global__ void InformationFilterUpdate_kernel2(signed char ipiv[16]);
static __global__ void InformationFilterUpdate_kernel3(const signed char Ik[256],
  double b_Ik[256]);
static __global__ void InformationFilterUpdate_kernel4(double Rw[64]);
static __global__ void InformationFilterUpdate_kernel5(const double Rw[8],
  double b_Rw[64]);
static __global__ void InformationFilterUpdate_kernel6(double Rw_inv[64]);
static __global__ void InformationFilterUpdate_kernel7(signed char ipiv[8]);
static __global__ void InformationFilterUpdate_kernel8(signed char p[8]);
static __global__ void InformationFilterUpdate_kernel9(double Fk_inv[256]);
static __global__ void ab_InformationFilterUpdate_kern(const double H_linear[208],
  double A[208]);
static __global__ void b_InformationFilterUpdate_kerne(const double T, double
  Pk_init[256]);
static void b_rand(double r[256]);
static __global__ void bb_InformationFilterUpdate_kern(const double xk_m_out[16],
  const double H_linear[208], const double hk[13], const double y_meas[13],
  double b_y_meas[13]);
static __global__ void c_InformationFilterUpdate_kerne(const signed char iv[16],
  const signed char iv1[16], const signed char iv2[16], const signed char iv3[16],
  const signed char iv4[16], const signed char iv5[16], const signed char iv6[16],
  const signed char iv7[16], signed char ipiv[16], double Pk_init[256]);
static __global__ void cb_InformationFilterUpdate_kern(const double y_meas[13],
  const double C[208], double op[16]);
static __global__ void d_InformationFilterUpdate_kerne(signed char p[16]);
static __global__ void e_InformationFilterUpdate_kerne(const double Fk_inv[256],
  double Pk_init[256]);
static void eml_rand_init();
static void eml_rand_mcg16807_stateful_init();
static void eml_rand_shr3cong_stateful_init();
static __global__ void f_InformationFilterUpdate_kerne(const double Gk[128],
  double A[128]);
static __global__ void g_InformationFilterUpdate_kerne(const double Rw_inv[64],
  double Rw[64], double y[64]);
static __global__ void h_InformationFilterUpdate_kerne(signed char ipiv[8]);
static __global__ void i_InformationFilterUpdate_kerne(signed char p[8]);
static __global__ void j_InformationFilterUpdate_kerne(const double Gk[128],
  double A[128]);
static __global__ void k_InformationFilterUpdate_kerne(double ih[16]);
static __global__ void l_InformationFilterUpdate_kerne(const double ih[16],
  const double Ik[256], double b_Ik[16]);
static __global__ void m_InformationFilterUpdate_kerne(const double Ik[16],
  const double Fk_inv[256], double ih[16]);
static __global__ void n_InformationFilterUpdate_kerne(const double Pk_init[256],
  const double ih[16], double op[16]);
static __global__ void o_InformationFilterUpdate_kerne(const double C[256],
  double Ih[256], double Pk_init[256]);
static __global__ void p_InformationFilterUpdate_kerne(signed char ipiv[16]);
static __global__ void q_InformationFilterUpdate_kerne(signed char p[16]);
static __global__ void r_InformationFilterUpdate_kerne(const double op[16],
  const double Pk_init[256], double xk_m_out[16]);
static __global__ void s_InformationFilterUpdate_kerne(const double
  B_usedMeas_vec[13], const double Re[13], double Re_inv[13]);
static __global__ void t_InformationFilterUpdate_kerne(double Re_inv[169]);
static __global__ void u_InformationFilterUpdate_kerne(const double Re_inv[13],
  double b_Re_inv[169]);
static __global__ void v_InformationFilterUpdate_kerne(double H_linear[208]);
static __global__ void w_InformationFilterUpdate_kerne(const signed char iv8[16],
  const signed char iv9[16], const signed char iv10[16], const signed char iv11
  [16], double H_linear[208]);
static __global__ void x_InformationFilterUpdate_kerne(const double xk_m_out[16],
  const double H_linear[208], double hk[13]);
static __global__ void y_InformationFilterUpdate_kerne(const double smax, const
  double delta, const double L_imuToRear, const double xk_m_out[16], double
  H_linear[208], double hk[13]);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Pk_init[256]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void InformationFilterUpdate_kernel1
  (double Pk_init[256])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 256) {
    Pk_init[k] = floor(Pk_init[k] * 2.0);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel2
  (signed char ipiv[16])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    // cinclude comment
    // cinclude comment
    ipiv[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char Ik[256]
//                double b_Ik[256]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void InformationFilterUpdate_kernel3
  (const signed char Ik[256], double b_Ik[256])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 256) {
    // cinclude comment
    b_Ik[k] = static_cast<double>(Ik[k]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void InformationFilterUpdate_kernel4
  (double Rw[64])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 64) {
    // 4*4
    // ---- Prediction step -------------------------
    Rw[k] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw[8]
//                double b_Rw[64]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel5(
  const double Rw[8], double b_Rw[64])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 8) {
    b_Rw[j + (j << 3)] = Rw[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw_inv[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void InformationFilterUpdate_kernel6
  (double Rw_inv[64])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 64) {
    Rw_inv[k] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel7
  (signed char ipiv[8])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 8) {
    ipiv[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel8
  (signed char p[8])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 8) {
    p[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Fk_inv[256]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void InformationFilterUpdate_kernel9
  (double Fk_inv[256])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 256) {
    //  System matrix
    Fk_inv[k] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double H_linear[208]
//                double A[208]
// Return Type  : void
//
static __global__ __launch_bounds__(224, 1) void ab_InformationFilterUpdate_kern
  (const double H_linear[208], double A[208])
{
  unsigned int threadId;
  int k;
  int i3;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i3 = static_cast<int>(threadId % 16U);
  k = static_cast<int>((threadId - static_cast<unsigned int>(i3)) / 16U);
  if (k < 13) {
    A[i3 + (k << 4)] = H_linear[k + 13 * i3];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double T
//                double Pk_init[256]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void b_InformationFilterUpdate_kerne(
  const double T, double Pk_init[256])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Pk_init[110] = 0.0;
    Pk_init[126] = 0.0;
    Pk_init[142] = 0.0;
    Pk_init[158] = 0.0;
    Pk_init[174] = 0.0;
    Pk_init[190] = 0.0;
    Pk_init[206] = 0.0;
    Pk_init[222] = 0.0;
    Pk_init[238] = 1.0;
    Pk_init[254] = T;
  }
}

//
// Arguments    : double r[256]
// Return Type  : void
//
static void b_rand(double r[256])
{
  unsigned int y;
  unsigned int u[2];
  if (method == 4U) {
    for (int k = 0; k < 256; k++) {
      int hi;
      unsigned int b_r;
      hi = static_cast<int>(state / 127773U);
      b_r = 16807U * (state - hi * 127773U);
      y = 2836U * hi;
      if (b_r < y) {
        state = ~(y - b_r) & 2147483647U;
      } else {
        state = b_r - y;
      }

      r[k] = static_cast<double>(state) * 4.6566128752457969E-10;
    }
  } else if (method == 5U) {
    for (int k = 0; k < 256; k++) {
      unsigned int b_r;
      b_r = 69069U * b_state[0] + 1234567U;
      y = b_state[1] ^ b_state[1] << 13;
      y ^= y >> 17;
      y ^= y << 5;
      b_state[0] = b_r;
      b_state[1] = y;
      r[k] = static_cast<double>(b_r + y) * 2.328306436538696E-10;
    }
  } else {
    int hi;
    unsigned int b_r;
    if (!state_not_empty) {
      for (hi = 0; hi < 625; hi++) {
        c_state[hi] = 0U;
      }

      b_r = 5489U;
      c_state[0] = 5489U;
      for (hi = 0; hi < 623; hi++) {
        b_r = ((b_r ^ b_r >> 30U) * 1812433253U + hi) + 1U;
        c_state[hi + 1] = b_r;
      }

      c_state[624] = 624U;
      state_not_empty = true;
    }

    for (int k = 0; k < 256; k++) {
      double c_r;

      // ========================= COPYRIGHT NOTICE ============================ 
      //  This is a uniform (0,1) pseudorandom number generator based on:        
      //                                                                         
      //  A C-program for MT19937, with initialization improved 2002/1/26.       
      //  Coded by Takuji Nishimura and Makoto Matsumoto.                        
      //                                                                         
      //  Copyright (C) 1997 - 2002, Makoto Matsumoto and Takuji Nishimura,      
      //  All rights reserved.                                                   
      //                                                                         
      //  Redistribution and use in source and binary forms, with or without     
      //  modification, are permitted provided that the following conditions     
      //  are met:                                                               
      //                                                                         
      //    1. Redistributions of source code must retain the above copyright    
      //       notice, this list of conditions and the following disclaimer.     
      //                                                                         
      //    2. Redistributions in binary form must reproduce the above copyright 
      //       notice, this list of conditions and the following disclaimer      
      //       in the documentation and/or other materials provided with the     
      //       distribution.                                                     
      //                                                                         
      //    3. The names of its contributors may not be used to endorse or       
      //       promote products derived from this software without specific      
      //       prior written permission.                                         
      //                                                                         
      //  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS    
      //  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT      
      //  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR  
      //  A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT  
      //  OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,  
      //  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT       
      //  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,  
      //  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY  
      //  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT    
      //  (INCLUDING  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE 
      //  OF THIS  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  
      //                                                                         
      // =============================   END   ================================= 
      do {
        for (hi = 0; hi < 2; hi++) {
          b_r = c_state[624] + 1U;
          if (b_r >= 625U) {
            int kk;
            for (kk = 0; kk < 227; kk++) {
              y = (c_state[kk] & 2147483648U) | (c_state[kk + 1] & 2147483647U);
              if ((y & 1U) == 0U) {
                y >>= 1U;
              } else {
                y = y >> 1U ^ 2567483615U;
              }

              c_state[kk] = c_state[kk + 397] ^ y;
            }

            for (kk = 0; kk < 396; kk++) {
              y = (c_state[kk + 227] & 2147483648U) | (c_state[kk + 228] &
                2147483647U);
              if ((y & 1U) == 0U) {
                y >>= 1U;
              } else {
                y = y >> 1U ^ 2567483615U;
              }

              c_state[kk + 227] = c_state[kk] ^ y;
            }

            y = (c_state[623] & 2147483648U) | (c_state[0] & 2147483647U);
            if ((y & 1U) == 0U) {
              y >>= 1U;
            } else {
              y = y >> 1U ^ 2567483615U;
            }

            c_state[623] = c_state[396] ^ y;
            b_r = 1U;
          }

          y = c_state[static_cast<int>(b_r) - 1];
          c_state[624] = b_r;
          y ^= y >> 11U;
          y ^= y << 7U & 2636928640U;
          y ^= y << 15U & 4022730752U;
          y ^= y >> 18U;
          u[hi] = y;
        }

        u[0] >>= 5U;
        u[1] >>= 6U;
        c_r = 1.1102230246251565E-16 * (static_cast<double>(u[0]) * 6.7108864E+7
          + static_cast<double>(u[1]));
      } while (c_r == 0.0);

      r[k] = c_r;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double xk_m_out[16]
//                const double H_linear[208]
//                const double hk[13]
//                const double y_meas[13]
//                double b_y_meas[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void bb_InformationFilterUpdate_kern(
  const double xk_m_out[16], const double H_linear[208], const double hk[13],
  const double y_meas[13], double b_y_meas[13])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 13) {
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += H_linear[k + 13 * i3] * xk_m_out[i3];
    }

    b_y_meas[k] = (y_meas[k] - hk[k]) + d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv[16]
//                const signed char iv1[16]
//                const signed char iv2[16]
//                const signed char iv3[16]
//                const signed char iv4[16]
//                const signed char iv5[16]
//                const signed char iv6[16]
//                const signed char iv7[16]
//                signed char ipiv[16]
//                double Pk_init[256]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void c_InformationFilterUpdate_kerne(
  const signed char iv[16], const signed char iv1[16], const signed char iv2[16],
  const signed char iv3[16], const signed char iv4[16], const signed char iv5[16],
  const signed char iv6[16], const signed char iv7[16], signed char ipiv[16],
  double Pk_init[256])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    // cinclude comment
    Pk_init[(k << 4) + 1] = static_cast<double>(iv7[k]);
    Pk_init[(k << 4) + 3] = static_cast<double>(iv6[k]);
    Pk_init[(k << 4) + 5] = static_cast<double>(iv5[k]);
    Pk_init[(k << 4) + 7] = static_cast<double>(iv4[k]);
    Pk_init[(k << 4) + 9] = static_cast<double>(iv3[k]);
    Pk_init[(k << 4) + 11] = static_cast<double>(iv2[k]);
    Pk_init[(k << 4) + 13] = static_cast<double>(iv1[k]);
    Pk_init[(k << 4) + 15] = static_cast<double>(iv[k]);
    ipiv[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double y_meas[13]
//                const double C[208]
//                double op[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void cb_InformationFilterUpdate_kern(
  const double y_meas[13], const double C[208], double op[16])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    d = 0.0;
    for (int i3 = 0; i3 < 13; i3++) {
      d += C[k + (i3 << 4)] * y_meas[i3];
    }

    op[k] += d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void d_InformationFilterUpdate_kerne
  (signed char p[16])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    p[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Fk_inv[256]
//                double Pk_init[256]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void e_InformationFilterUpdate_kerne
  (const double Fk_inv[256], double Pk_init[256])
{
  unsigned int threadId;
  int k;
  int i3;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i3 = static_cast<int>(threadId % 16U);
  k = static_cast<int>((threadId - static_cast<unsigned int>(i3)) / 16U);
  if (k < 16) {
    Pk_init[i3 + (k << 4)] = Fk_inv[k + (i3 << 4)];
  }
}

//
// Arguments    : void
// Return Type  : void
//
static void eml_rand_init()
{
  method = 7U;
}

//
// Arguments    : void
// Return Type  : void
//
static void eml_rand_mcg16807_stateful_init()
{
  state = 1144108930U;
}

//
// Arguments    : void
// Return Type  : void
//
static void eml_rand_shr3cong_stateful_init()
{
  for (int i = 0; i < 2; i++) {
    b_state[i] = 158852560U * i + 362436069U;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[128]
//                double A[128]
// Return Type  : void
//
static __global__ __launch_bounds__(128, 1) void f_InformationFilterUpdate_kerne
  (const double Gk[128], double A[128])
{
  unsigned int threadId;
  int k;
  int i3;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i3 = static_cast<int>(threadId % 8U);
  k = static_cast<int>((threadId - static_cast<unsigned int>(i3)) / 8U);
  if (k < 16) {
    A[i3 + (k << 3)] = Gk[k + (i3 << 4)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw_inv[64]
//                double Rw[64]
//                double y[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void g_InformationFilterUpdate_kerne(
  const double Rw_inv[64], double Rw[64], double y[64])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 64) {
    y[k] = 0.0;
    Rw[k] += Rw_inv[k];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void h_InformationFilterUpdate_kerne
  (signed char ipiv[8])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 8) {
    ipiv[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void i_InformationFilterUpdate_kerne
  (signed char p[8])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 8) {
    p[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[128]
//                double A[128]
// Return Type  : void
//
static __global__ __launch_bounds__(128, 1) void j_InformationFilterUpdate_kerne
  (const double Gk[128], double A[128])
{
  unsigned int threadId;
  int k;
  int i3;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i3 = static_cast<int>(threadId % 8U);
  k = static_cast<int>((threadId - static_cast<unsigned int>(i3)) / 8U);
  if (k < 16) {
    A[i3 + (k << 3)] = Gk[k + (i3 << 4)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double ih[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void k_InformationFilterUpdate_kerne
  (double ih[16])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    ih[13] = 0.0;
    ih[14] = 0.0;
    ih[15] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ih[16]
//                const double Ik[256]
//                double b_Ik[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void l_InformationFilterUpdate_kerne(
  const double ih[16], const double Ik[256], double b_Ik[16])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += Ik[k + (i3 << 4)] * ih[i3];
    }

    b_Ik[k] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Ik[16]
//                const double Fk_inv[256]
//                double ih[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void m_InformationFilterUpdate_kerne(
  const double Ik[16], const double Fk_inv[256], double ih[16])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += Fk_inv[i3 + (k << 4)] * Ik[i3];
    }

    ih[k] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Pk_init[256]
//                const double ih[16]
//                double op[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void n_InformationFilterUpdate_kerne(
  const double Pk_init[256], const double ih[16], double op[16])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    // 4*1
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += Pk_init[k + (i3 << 4)] * ih[i3];
    }

    op[k] = ih[k] - d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double C[256]
//                double Ih[256]
//                double Pk_init[256]
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void o_InformationFilterUpdate_kerne
  (const double C[256], double Ih[256], double Pk_init[256])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 256) {
    // 4*1
    Pk_init[k] = 0.0;
    Ih[k] -= C[k];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void p_InformationFilterUpdate_kerne
  (signed char ipiv[16])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    // cinclude comment
    ipiv[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void q_InformationFilterUpdate_kerne
  (signed char p[16])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    p[k] = static_cast<signed char>(k + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double op[16]
//                const double Pk_init[256]
//                double xk_m_out[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void r_InformationFilterUpdate_kerne(
  const double op[16], const double Pk_init[256], double xk_m_out[16])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    // 4*4
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += Pk_init[k + (i3 << 4)] * op[i3];
    }

    xk_m_out[k] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double B_usedMeas_vec[13]
//                const double Re[13]
//                double Re_inv[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void s_InformationFilterUpdate_kerne(
  const double B_usedMeas_vec[13], const double Re[13], double Re_inv[13])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 13) {
    // 1*1
    Re_inv[k] = 1.0 / Re[k] * B_usedMeas_vec[k];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void t_InformationFilterUpdate_kerne
  (double Re_inv[169])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 169) {
    // 1*13
    Re_inv[k] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Re_inv[13]
//                double b_Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void u_InformationFilterUpdate_kerne(
  const double Re_inv[13], double b_Re_inv[169])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 13) {
    b_Re_inv[j + 13 * j] = Re_inv[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double H_linear[208]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void v_InformationFilterUpdate_kerne
  (double H_linear[208])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    H_linear[205] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv8[16]
//                const signed char iv9[16]
//                const signed char iv10[16]
//                const signed char iv11[16]
//                double H_linear[208]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void w_InformationFilterUpdate_kerne(
  const signed char iv8[16], const signed char iv9[16], const signed char iv10
  [16], const signed char iv11[16], double H_linear[208])
{
  unsigned int threadId;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 16) {
    H_linear[13 * k] = static_cast<double>(iv11[k]);
    H_linear[13 * k + 2] = static_cast<double>(iv10[k]);
    H_linear[13 * k + 3] = static_cast<double>(iv9[k]);
    H_linear[13 * k + 4] = static_cast<double>(iv8[k]);
    H_linear[13 * k + 5] = static_cast<double>(iv8[k]);
    H_linear[13 * k + 11] = static_cast<double>(iv9[k]);
    H_linear[13 * k + 12] = static_cast<double>(iv9[k]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double xk_m_out[16]
//                const double H_linear[208]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void x_InformationFilterUpdate_kerne(
  const double xk_m_out[16], const double H_linear[208], double hk[13])
{
  unsigned int threadId;
  double d;
  int k;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  k = static_cast<int>(threadId);
  if (k < 13) {
    //  Nonlinear parts
    d = 0.0;
    for (int i3 = 0; i3 < 16; i3++) {
      d += H_linear[k + 13 * i3] * xk_m_out[i3];
    }

    hk[k] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double smax
//                const double delta
//                const double L_imuToRear
//                const double xk_m_out[16]
//                double H_linear[208]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void y_InformationFilterUpdate_kerne(
  const double smax, const double delta, const double L_imuToRear, const double
  xk_m_out[16], double H_linear[208], double hk[13])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    hk[0] -= xk_m_out[2] * xk_m_out[2] * L_imuToRear;
    hk[1] += xk_m_out[0] * xk_m_out[2];
    hk[10] = xk_m_out[0] * cos(delta) + smax * xk_m_out[2] * sin(delta);

    // updates the Hk 13*4matrix HK(5,1)=Hk matrix's 5th row 1st element
    // being updated with cos(delta)value . Likewise for all.
    H_linear[26] = -2.0 * xk_m_out[2] * L_imuToRear;
    H_linear[1] = xk_m_out[2];
    H_linear[27] = xk_m_out[0];
    H_linear[4] = cos(delta);
    H_linear[10] = cos(delta);
    H_linear[36] = smax * sin(delta);
  }
}

//
// persistent Ik ik
// Arguments    : const double y_meas[13]
//                const double B_usedMeas_vec[13]
//                const double initialization_vec[4]
//                double delta
//                const double Rw[8]
//                const double Re[13]
//                double L_imuToRear
//                double L_geometricWheelbase
//                const double L_trackWidth[5]
//                const double L_axlePos[5]
//                double T
//                double xk_m_out[16]
//                double op[16]
// Return Type  : void
//
void InformationFilterUpdate(const double y_meas[13], const double
  B_usedMeas_vec[13], const double initialization_vec[4], double delta, const
  double Rw[8], const double Re[13], double L_imuToRear, double
  L_geometricWheelbase, const double L_trackWidth[5], const double L_axlePos[5],
  double T, double xk_m_out[16], double op[16])
{
  int j;
  int c;
  int jp1j;
  int b_c;
  int iy;
  int i;
  static const signed char Ik[256] = { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 1 };

  int ix;
  double smax;
  signed char b_i;
  int k;
  double s;
  int jy;
  int i1;
  int ia;
  int i2;
  static const signed char iv[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1 };

  static const signed char iv1[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    0, 0 };

  static const signed char iv2[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0,
    0, 0 };

  static const signed char iv3[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0,
    0, 0 };

  static const signed char iv4[16] = { 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0,
    0, 0 };

  static const signed char iv5[16] = { 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0 };

  static const signed char iv6[16] = { 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0 };

  static const signed char iv7[16] = { 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0 };

  int ic;
  double C[128];
  double b_C[128];
  double c_C[128];
  static const signed char iv8[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0,
    0, 0 };

  static const signed char iv9[16] = { 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0,
    0, 0 };

  static const signed char iv10[16] = { 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0,
    1, 0 };

  static const signed char iv11[16] = { 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1,
    0, 0 };

  double (*gpu_Pk_init)[256];
  signed char (*gpu_ipiv)[16];
  signed char (*gpu_Ik)[256];
  double (*b_gpu_Ik)[256];
  double (*gpu_Rw)[64];
  double (*b_gpu_Rw)[8];
  double (*gpu_Rw_inv)[64];
  signed char (*b_gpu_ipiv)[8];
  signed char (*gpu_p)[8];
  double (*gpu_Fk_inv)[256];
  signed char (*gpu_iv)[16];
  signed char (*gpu_iv1)[16];
  signed char (*gpu_iv2)[16];
  signed char (*gpu_iv3)[16];
  signed char (*gpu_iv4)[16];
  signed char (*gpu_iv5)[16];
  signed char (*gpu_iv6)[16];
  signed char (*gpu_iv7)[16];
  signed char (*b_gpu_p)[16];
  double (*gpu_Gk)[128];
  double (*gpu_A)[128];
  double (*gpu_y)[64];
  double (*gpu_ih)[16];
  double (*c_gpu_Ik)[16];
  double (*gpu_op)[16];
  double (*gpu_C)[256];
  double (*gpu_Ih)[256];
  double (*gpu_xk_m_out)[16];
  double (*gpu_B_usedMeas_vec)[13];
  double (*gpu_Re)[13];
  double (*gpu_Re_inv)[13];
  double (*b_gpu_Re_inv)[169];
  double (*gpu_H_linear)[208];
  signed char (*gpu_iv8)[16];
  signed char (*gpu_iv9)[16];
  signed char (*gpu_iv10)[16];
  signed char (*gpu_iv11)[16];
  double (*gpu_hk)[13];
  double (*b_gpu_A)[208];
  double (*gpu_y_meas)[13];
  double (*b_gpu_y_meas)[13];
  double (*b_gpu_C)[208];
  boolean_T syncIsDirty;
  if (!isInitialized_InformationFilterUpdate) {
    InformationFilterUpdate_initialize();
  }

  hipMallocManaged(&b_gpu_C, 1664ULL);
  hipMallocManaged(&b_gpu_y_meas, 104ULL);
  hipMallocManaged(&b_gpu_A, 1664ULL);
  hipMallocManaged(&gpu_hk, 104ULL);
  hipMallocManaged(&gpu_H_linear, 1664ULL);
  hipMallocManaged(&b_gpu_Re_inv, 1352ULL);
  hipMallocManaged(&gpu_Re_inv, 104ULL);
  hipMallocManaged(&gpu_Ih, 2048ULL);
  hipMallocManaged(&gpu_C, 2048ULL);
  hipMallocManaged(&c_gpu_Ik, 128ULL);
  hipMallocManaged(&gpu_ih, 128ULL);
  hipMallocManaged(&gpu_y, 512ULL);
  hipMallocManaged(&gpu_A, 1024ULL);
  hipMallocManaged(&gpu_Gk, 1024ULL);
  hipMallocManaged(&b_gpu_p, 16ULL);
  hipMallocManaged(&gpu_Fk_inv, 2048ULL);
  hipMallocManaged(&gpu_p, 8ULL);
  hipMallocManaged(&b_gpu_ipiv, 8ULL);
  hipMallocManaged(&gpu_Rw_inv, 512ULL);
  hipMallocManaged(&gpu_Rw, 512ULL);
  hipMallocManaged(&b_gpu_Ik, 2048ULL);
  hipMallocManaged(&gpu_ipiv, 16ULL);
  hipMallocManaged(&gpu_Pk_init, 2048ULL);
  hipMallocManaged(&gpu_xk_m_out, 128ULL);
  hipMallocManaged(&gpu_op, 128ULL);
  hipMallocManaged(&gpu_Ik, 256ULL);
  hipMallocManaged(&b_gpu_Rw, 64ULL);
  hipMallocManaged(&gpu_iv, 16ULL);
  hipMallocManaged(&gpu_iv1, 16ULL);
  hipMallocManaged(&gpu_iv2, 16ULL);
  hipMallocManaged(&gpu_iv3, 16ULL);
  hipMallocManaged(&gpu_iv4, 16ULL);
  hipMallocManaged(&gpu_iv5, 16ULL);
  hipMallocManaged(&gpu_iv6, 16ULL);
  hipMallocManaged(&gpu_iv7, 16ULL);
  hipMallocManaged(&gpu_B_usedMeas_vec, 104ULL);
  hipMallocManaged(&gpu_Re, 104ULL);
  hipMallocManaged(&gpu_iv8, 16ULL);
  hipMallocManaged(&gpu_iv9, 16ULL);
  hipMallocManaged(&gpu_iv10, 16ULL);
  hipMallocManaged(&gpu_iv11, 16ULL);
  hipMallocManaged(&gpu_y_meas, 104ULL);
  hipMemcpy(gpu_y_meas, (void *)&y_meas[0], 104ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv11, (void *)&iv11[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv10, (void *)&iv10[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv9, (void *)&iv9[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv8, (void *)&iv8[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_Re, (void *)&Re[0], 104ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B_usedMeas_vec, (void *)&B_usedMeas_vec[0], 104ULL,
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv7, (void *)&iv7[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv6, (void *)&iv6[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv5, (void *)&iv5[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv4, (void *)&iv4[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv3, (void *)&iv3[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv2, (void *)&iv2[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv1, (void *)&iv1[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv, (void *)&iv[0], 16ULL, hipMemcpyHostToDevice);
  hipMemcpy(b_gpu_Rw, (void *)&Rw[0], 64ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_Ik, (void *)&Ik[0], 256ULL, hipMemcpyHostToDevice);
  b_rand(*gpu_Pk_init);
  InformationFilterUpdate_kernel1<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_Pk_init);
  InformationFilterUpdate_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 15; j++) {
    c = j * 17;
    jp1j = c - 12;
    b_c = 14 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Pk_init)[c]);
    for (k = 0; k <= b_c; k++) {
      ix++;
      s = std::abs((*gpu_Pk_init)[ix]);
      if (s > smax) {
        iy = k + 1;
        smax = s;
      }
    }

    if ((*gpu_Pk_init)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        b_c = iy;
        for (k = 0; k < 16; k++) {
          ix = j + k * 16;
          iy = b_c + k * 16;
          smax = (*gpu_Pk_init)[ix];
          (*gpu_Pk_init)[ix] = (*gpu_Pk_init)[iy];
          (*gpu_Pk_init)[iy] = smax;
        }
      }

      i1 = (c - j) + 2;
      for (i = 0; i <= i1 - jp1j; i++) {
        iy = (c + i) + 1;
        (*gpu_Pk_init)[iy] /= (*gpu_Pk_init)[c];
      }
    }

    b_c = 14 - j;
    iy = c + 18;
    jy = c + 16;
    for (ia = 0; ia <= b_c; ia++) {
      smax = (*gpu_Pk_init)[jy];
      if ((*gpu_Pk_init)[jy] != 0.0) {
        ix = c;
        i1 = iy - 14;
        i2 = iy - j;
        for (jp1j = 0; jp1j <= i2 - i1; jp1j++) {
          i = (iy + jp1j) - 1;
          (*gpu_Pk_init)[i] += (*gpu_Pk_init)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 16;
      iy += 16;
    }
  }

  InformationFilterUpdate_kernel3<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_Ik, *b_gpu_Ik);
  syncIsDirty = true;
  for (i = 0; i < 15; i++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    b_i = (*gpu_ipiv)[i];
    if ((*gpu_ipiv)[i] != i + 1) {
      for (j = 0; j < 16; j++) {
        iy = static_cast<int>((*b_gpu_Ik)[i + (j << 4)]);
        (*b_gpu_Ik)[i + (j << 4)] = (*b_gpu_Ik)[(b_i + (j << 4)) - 1];
        (*b_gpu_Ik)[(b_i + (j << 4)) - 1] = iy;
      }
    }
  }

  for (j = 0; j < 16; j++) {
    jp1j = j << 4;
    for (k = 0; k < 16; k++) {
      jy = k << 4;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*b_gpu_Ik)[k + jp1j] != 0.0) {
        for (i = 0; i <= 14 - k; i++) {
          iy = (k + i) + 1;
          (*b_gpu_Ik)[iy + jp1j] -= (*b_gpu_Ik)[k + jp1j] * (*gpu_Pk_init)[iy +
            jy];
        }
      }
    }
  }

  for (j = 0; j < 16; j++) {
    jp1j = (j << 4) - 1;
    for (k = 0; k < 16; k++) {
      iy = 16 - k;
      jy = (15 - k) << 4;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*b_gpu_Ik)[(jp1j - k) + 16] != 0.0) {
        (*b_gpu_Ik)[(jp1j - k) + 16] /= (*gpu_Pk_init)[(jy - k) + 15];
        for (i = 0; i <= iy - 2; i++) {
          (*b_gpu_Ik)[(i + jp1j) + 1] -= (*b_gpu_Ik)[(jp1j - k) + 16] *
            (*gpu_Pk_init)[i + jy];
        }
      }
    }
  }

  // 4*4
  // ---- Prediction step -------------------------
  InformationFilterUpdate_kernel4<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Rw);
  InformationFilterUpdate_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_Rw, *gpu_Rw);
  InformationFilterUpdate_kernel6<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Rw_inv);
  InformationFilterUpdate_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 7; j++) {
    c = j * 9;
    jp1j = c - 4;
    b_c = 6 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Rw)[c]);
    for (k = 0; k <= b_c; k++) {
      ix++;
      s = std::abs((*gpu_Rw)[ix]);
      if (s > smax) {
        iy = k + 1;
        smax = s;
      }
    }

    if ((*gpu_Rw)[c + iy] != 0.0) {
      if (iy != 0) {
        (*b_gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        b_c = iy;
        for (k = 0; k < 8; k++) {
          ix = j + k * 8;
          iy = b_c + k * 8;
          smax = (*gpu_Rw)[ix];
          (*gpu_Rw)[ix] = (*gpu_Rw)[iy];
          (*gpu_Rw)[iy] = smax;
        }
      }

      i1 = (c - j) + 2;
      for (i = 0; i <= i1 - jp1j; i++) {
        iy = (c + i) + 1;
        (*gpu_Rw)[iy] /= (*gpu_Rw)[c];
      }
    }

    b_c = 6 - j;
    iy = c + 10;
    jy = c + 8;
    for (ia = 0; ia <= b_c; ia++) {
      smax = (*gpu_Rw)[jy];
      if ((*gpu_Rw)[jy] != 0.0) {
        ix = c;
        i1 = iy - 6;
        i2 = iy - j;
        for (jp1j = 0; jp1j <= i2 - i1; jp1j++) {
          i = (iy + jp1j) - 1;
          (*gpu_Rw)[i] += (*gpu_Rw)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 8;
      iy += 8;
    }
  }

  InformationFilterUpdate_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 7; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*b_gpu_ipiv)[k] > k + 1) {
      iy = (*gpu_p)[(*b_gpu_ipiv)[k] - 1];
      (*gpu_p)[(*b_gpu_ipiv)[k] - 1] = (*gpu_p)[k];
      (*gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 8; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    b_i = (*gpu_p)[k];
    (*gpu_Rw_inv)[k + (((*gpu_p)[k] - 1) << 3)] = 1.0;
    for (j = 0; j <= 7 - k; j++) {
      ia = k + j;
      if ((*gpu_Rw_inv)[ia + ((b_i - 1) << 3)] != 0.0) {
        for (i = 0; i <= 6 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_Rw_inv)[iy + ((b_i - 1) << 3)] -= (*gpu_Rw_inv)[ia + ((b_i - 1) <<
            3)] * (*gpu_Rw)[iy + (ia << 3)];
        }
      }
    }
  }

  for (j = 0; j < 8; j++) {
    jp1j = (j << 3) - 1;
    for (k = 0; k < 8; k++) {
      iy = 8 - k;
      jy = (7 - k) << 3;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Rw_inv)[(jp1j - k) + 8] != 0.0) {
        (*gpu_Rw_inv)[(jp1j - k) + 8] /= (*gpu_Rw)[(jy - k) + 7];
        for (i = 0; i <= iy - 2; i++) {
          (*gpu_Rw_inv)[(i + jp1j) + 1] -= (*gpu_Rw_inv)[(jp1j - k) + 8] *
            (*gpu_Rw)[i + jy];
        }
      }
    }
  }

  //  System matrix
  InformationFilterUpdate_kernel9<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_Fk_inv);
  hipDeviceSynchronize();
  (*gpu_Pk_init)[0] = 1.0;
  (*gpu_Pk_init)[16] = T;
  (*gpu_Pk_init)[32] = 0.0;
  (*gpu_Pk_init)[48] = 0.0;
  (*gpu_Pk_init)[64] = 0.0;
  (*gpu_Pk_init)[80] = 0.0;
  (*gpu_Pk_init)[96] = 0.0;
  (*gpu_Pk_init)[112] = 0.0;
  (*gpu_Pk_init)[128] = 0.0;
  (*gpu_Pk_init)[144] = 0.0;
  (*gpu_Pk_init)[160] = 0.0;
  (*gpu_Pk_init)[176] = 0.0;
  (*gpu_Pk_init)[192] = 0.0;
  (*gpu_Pk_init)[208] = 0.0;
  (*gpu_Pk_init)[224] = 0.0;
  (*gpu_Pk_init)[240] = 0.0;
  (*gpu_Pk_init)[2] = 0.0;
  (*gpu_Pk_init)[18] = 0.0;
  (*gpu_Pk_init)[34] = 1.0;
  (*gpu_Pk_init)[50] = 0.0;
  (*gpu_Pk_init)[66] = 0.0;
  (*gpu_Pk_init)[82] = 0.0;
  (*gpu_Pk_init)[98] = 0.0;
  (*gpu_Pk_init)[114] = 0.0;
  (*gpu_Pk_init)[130] = T;
  (*gpu_Pk_init)[146] = 0.0;
  (*gpu_Pk_init)[162] = 0.0;
  (*gpu_Pk_init)[178] = 0.0;
  (*gpu_Pk_init)[194] = 0.0;
  (*gpu_Pk_init)[210] = 0.0;
  (*gpu_Pk_init)[226] = 0.0;
  (*gpu_Pk_init)[242] = 0.0;
  (*gpu_Pk_init)[4] = 0.0;
  (*gpu_Pk_init)[20] = 0.0;
  (*gpu_Pk_init)[36] = 0.0;
  (*gpu_Pk_init)[52] = 0.0;
  (*gpu_Pk_init)[68] = 1.0;
  (*gpu_Pk_init)[84] = 0.0;
  (*gpu_Pk_init)[100] = 0.0;
  (*gpu_Pk_init)[116] = 0.0;
  (*gpu_Pk_init)[132] = 0.0;
  (*gpu_Pk_init)[148] = 0.0;
  (*gpu_Pk_init)[164] = 0.0;
  (*gpu_Pk_init)[180] = 0.0;
  (*gpu_Pk_init)[196] = 0.0;
  (*gpu_Pk_init)[212] = 0.0;
  (*gpu_Pk_init)[228] = 0.0;
  (*gpu_Pk_init)[244] = T;
  (*gpu_Pk_init)[6] = 0.0;
  (*gpu_Pk_init)[22] = 0.0;
  (*gpu_Pk_init)[38] = 0.0;
  (*gpu_Pk_init)[54] = 0.0;
  (*gpu_Pk_init)[70] = 0.0;
  (*gpu_Pk_init)[86] = 0.0;
  (*gpu_Pk_init)[102] = 1.0;
  (*gpu_Pk_init)[118] = 0.0;
  (*gpu_Pk_init)[134] = T;
  (*gpu_Pk_init)[150] = 0.0;
  (*gpu_Pk_init)[166] = 0.0;
  (*gpu_Pk_init)[182] = 0.0;
  (*gpu_Pk_init)[198] = 0.0;
  (*gpu_Pk_init)[214] = 0.0;
  (*gpu_Pk_init)[230] = 0.0;
  (*gpu_Pk_init)[246] = 0.0;
  (*gpu_Pk_init)[8] = 0.0;
  (*gpu_Pk_init)[24] = 0.0;
  (*gpu_Pk_init)[40] = 0.0;
  (*gpu_Pk_init)[56] = 0.0;
  (*gpu_Pk_init)[72] = 0.0;
  (*gpu_Pk_init)[88] = 0.0;
  (*gpu_Pk_init)[104] = 0.0;
  (*gpu_Pk_init)[120] = 0.0;
  (*gpu_Pk_init)[136] = 1.0;
  (*gpu_Pk_init)[152] = 0.0;
  (*gpu_Pk_init)[168] = 0.0;
  (*gpu_Pk_init)[184] = T;
  (*gpu_Pk_init)[200] = 0.0;
  (*gpu_Pk_init)[216] = 0.0;
  (*gpu_Pk_init)[232] = 0.0;
  (*gpu_Pk_init)[248] = 0.0;
  (*gpu_Pk_init)[10] = 0.0;
  (*gpu_Pk_init)[26] = 0.0;
  (*gpu_Pk_init)[42] = 0.0;
  (*gpu_Pk_init)[58] = 0.0;
  (*gpu_Pk_init)[74] = 0.0;
  (*gpu_Pk_init)[90] = 0.0;
  (*gpu_Pk_init)[106] = 0.0;
  (*gpu_Pk_init)[122] = 0.0;
  (*gpu_Pk_init)[138] = 0.0;
  (*gpu_Pk_init)[154] = 0.0;
  (*gpu_Pk_init)[170] = 1.0;
  (*gpu_Pk_init)[186] = 0.0;
  (*gpu_Pk_init)[202] = 0.0;
  (*gpu_Pk_init)[218] = 0.0;
  (*gpu_Pk_init)[234] = T;
  (*gpu_Pk_init)[250] = 0.0;
  (*gpu_Pk_init)[12] = 0.0;
  (*gpu_Pk_init)[28] = 0.0;
  (*gpu_Pk_init)[44] = 0.0;
  (*gpu_Pk_init)[60] = 0.0;
  (*gpu_Pk_init)[76] = 0.0;
  (*gpu_Pk_init)[92] = 0.0;
  (*gpu_Pk_init)[108] = 0.0;
  (*gpu_Pk_init)[124] = 0.0;
  (*gpu_Pk_init)[140] = T;
  (*gpu_Pk_init)[156] = 0.0;
  (*gpu_Pk_init)[172] = 0.0;
  (*gpu_Pk_init)[188] = 0.0;
  (*gpu_Pk_init)[204] = 1.0;
  (*gpu_Pk_init)[220] = 0.0;
  (*gpu_Pk_init)[236] = 0.0;
  (*gpu_Pk_init)[252] = 0.0;
  (*gpu_Pk_init)[14] = 0.0;
  (*gpu_Pk_init)[30] = 0.0;
  (*gpu_Pk_init)[46] = 0.0;
  (*gpu_Pk_init)[62] = 0.0;
  (*gpu_Pk_init)[78] = 0.0;
  (*gpu_Pk_init)[94] = 0.0;
  b_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(T,
    *gpu_Pk_init);
  c_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv, *gpu_iv1, *gpu_iv2, *gpu_iv3, *gpu_iv4, *gpu_iv5, *gpu_iv6,
     *gpu_iv7, *gpu_ipiv, *gpu_Pk_init);
  syncIsDirty = true;
  for (j = 0; j < 15; j++) {
    c = j * 17;
    jp1j = c - 12;
    b_c = 14 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Pk_init)[c]);
    for (k = 0; k <= b_c; k++) {
      ix++;
      s = std::abs((*gpu_Pk_init)[ix]);
      if (s > smax) {
        iy = k + 1;
        smax = s;
      }
    }

    if ((*gpu_Pk_init)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        b_c = iy;
        for (k = 0; k < 16; k++) {
          ix = j + k * 16;
          iy = b_c + k * 16;
          smax = (*gpu_Pk_init)[ix];
          (*gpu_Pk_init)[ix] = (*gpu_Pk_init)[iy];
          (*gpu_Pk_init)[iy] = smax;
        }
      }

      i1 = (c - j) + 2;
      for (i = 0; i <= i1 - jp1j; i++) {
        iy = (c + i) + 1;
        (*gpu_Pk_init)[iy] /= (*gpu_Pk_init)[c];
      }
    }

    b_c = 14 - j;
    iy = c + 18;
    jy = c + 16;
    for (ia = 0; ia <= b_c; ia++) {
      smax = (*gpu_Pk_init)[jy];
      if ((*gpu_Pk_init)[jy] != 0.0) {
        ix = c;
        i1 = iy - 14;
        i2 = iy - j;
        for (jp1j = 0; jp1j <= i2 - i1; jp1j++) {
          i = (iy + jp1j) - 1;
          (*gpu_Pk_init)[i] += (*gpu_Pk_init)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 16;
      iy += 16;
    }
  }

  d_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 15; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[k] > k + 1) {
      iy = (*b_gpu_p)[(*gpu_ipiv)[k] - 1];
      (*b_gpu_p)[(*gpu_ipiv)[k] - 1] = (*b_gpu_p)[k];
      (*b_gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 16; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    b_i = (*b_gpu_p)[k];
    (*gpu_Fk_inv)[k + (((*b_gpu_p)[k] - 1) << 4)] = 1.0;
    for (j = 0; j <= 15 - k; j++) {
      ia = k + j;
      if ((*gpu_Fk_inv)[ia + ((b_i - 1) << 4)] != 0.0) {
        for (i = 0; i <= 14 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_Fk_inv)[iy + ((b_i - 1) << 4)] -= (*gpu_Fk_inv)[ia + ((b_i - 1) <<
            4)] * (*gpu_Pk_init)[iy + (ia << 4)];
        }
      }
    }
  }

  for (j = 0; j < 16; j++) {
    jp1j = (j << 4) - 1;
    for (k = 0; k < 16; k++) {
      iy = 16 - k;
      jy = (15 - k) << 4;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Fk_inv)[(jp1j - k) + 16] != 0.0) {
        (*gpu_Fk_inv)[(jp1j - k) + 16] /= (*gpu_Pk_init)[(jy - k) + 15];
        for (i = 0; i <= iy - 2; i++) {
          (*gpu_Fk_inv)[(i + jp1j) + 1] -= (*gpu_Fk_inv)[(jp1j - k) + 16] *
            (*gpu_Pk_init)[i + jy];
        }
      }
    }
  }

  // 4*4
  //  Noise matrix
  if (syncIsDirty) {
    hipDeviceSynchronize();
  }

  (*gpu_Gk)[0] = T * T / 2.0;
  (*gpu_Gk)[16] = 0.0;
  (*gpu_Gk)[32] = T * T / 2.0;
  (*gpu_Gk)[48] = 0.0;
  (*gpu_Gk)[64] = T * T / 2.0;
  (*gpu_Gk)[80] = 0.0;
  (*gpu_Gk)[96] = T * T / 2.0;
  (*gpu_Gk)[112] = 0.0;
  (*gpu_Gk)[1] = T;
  (*gpu_Gk)[17] = 0.0;
  (*gpu_Gk)[33] = T;
  (*gpu_Gk)[49] = 0.0;
  (*gpu_Gk)[65] = T * T / 2.0;
  (*gpu_Gk)[81] = 0.0;
  (*gpu_Gk)[97] = T * T / 2.0;
  (*gpu_Gk)[113] = 0.0;
  (*gpu_Gk)[2] = 0.0;
  (*gpu_Gk)[18] = T * T / 2.0;
  (*gpu_Gk)[34] = 0.0;
  (*gpu_Gk)[50] = T * T / 2.0;
  (*gpu_Gk)[66] = T * T / 2.0;
  (*gpu_Gk)[82] = 0.0;
  (*gpu_Gk)[98] = T * T / 2.0;
  (*gpu_Gk)[114] = 0.0;
  (*gpu_Gk)[3] = 0.0;
  (*gpu_Gk)[19] = T;
  (*gpu_Gk)[35] = 0.0;
  (*gpu_Gk)[51] = T;
  (*gpu_Gk)[67] = 0.0;
  (*gpu_Gk)[83] = T;
  (*gpu_Gk)[99] = 0.0;
  (*gpu_Gk)[115] = T;
  (*gpu_Gk)[4] = T * T / 2.0;
  (*gpu_Gk)[20] = 0.0;
  (*gpu_Gk)[36] = T * T / 2.0;
  (*gpu_Gk)[52] = 0.0;
  (*gpu_Gk)[68] = 0.0;
  (*gpu_Gk)[84] = T;
  (*gpu_Gk)[100] = 0.0;
  (*gpu_Gk)[116] = T;
  (*gpu_Gk)[5] = T;
  (*gpu_Gk)[21] = 0.0;
  (*gpu_Gk)[37] = T;
  (*gpu_Gk)[53] = 0.0;
  (*gpu_Gk)[69] = 0.0;
  (*gpu_Gk)[85] = T;
  (*gpu_Gk)[101] = 0.0;
  (*gpu_Gk)[117] = T;
  (*gpu_Gk)[6] = 0.0;
  (*gpu_Gk)[22] = T * T / 2.0;
  (*gpu_Gk)[38] = 0.0;
  (*gpu_Gk)[54] = T * T / 2.0;
  (*gpu_Gk)[70] = 0.0;
  (*gpu_Gk)[86] = T;
  (*gpu_Gk)[102] = 0.0;
  (*gpu_Gk)[118] = T;
  (*gpu_Gk)[7] = 0.0;
  (*gpu_Gk)[23] = T;
  (*gpu_Gk)[39] = 0.0;
  (*gpu_Gk)[55] = T;
  (*gpu_Gk)[71] = 0.0;
  (*gpu_Gk)[87] = T;
  (*gpu_Gk)[103] = 0.0;
  (*gpu_Gk)[119] = T;
  (*gpu_Gk)[8] = T * T / 2.0;
  (*gpu_Gk)[24] = 0.0;
  (*gpu_Gk)[40] = T * T / 2.0;
  (*gpu_Gk)[56] = 0.0;
  (*gpu_Gk)[72] = 0.0;
  (*gpu_Gk)[88] = T;
  (*gpu_Gk)[104] = 0.0;
  (*gpu_Gk)[120] = T;
  (*gpu_Gk)[9] = T;
  (*gpu_Gk)[25] = 0.0;
  (*gpu_Gk)[41] = T;
  (*gpu_Gk)[57] = 0.0;
  (*gpu_Gk)[73] = 0.0;
  (*gpu_Gk)[89] = T;
  (*gpu_Gk)[105] = 0.0;
  (*gpu_Gk)[121] = T;
  (*gpu_Gk)[10] = 0.0;
  (*gpu_Gk)[26] = T * T / 2.0;
  (*gpu_Gk)[42] = 0.0;
  (*gpu_Gk)[58] = T * T / 2.0;
  (*gpu_Gk)[74] = 0.0;
  (*gpu_Gk)[90] = T;
  (*gpu_Gk)[106] = 0.0;
  (*gpu_Gk)[122] = T;
  (*gpu_Gk)[11] = 0.0;
  (*gpu_Gk)[27] = T;
  (*gpu_Gk)[43] = 0.0;
  (*gpu_Gk)[59] = T;
  (*gpu_Gk)[75] = 0.0;
  (*gpu_Gk)[91] = T;
  (*gpu_Gk)[107] = 0.0;
  (*gpu_Gk)[123] = T;
  (*gpu_Gk)[12] = T * T / 2.0;
  (*gpu_Gk)[28] = 0.0;
  (*gpu_Gk)[44] = T * T / 2.0;
  (*gpu_Gk)[60] = 0.0;
  (*gpu_Gk)[76] = 0.0;
  (*gpu_Gk)[92] = T;
  (*gpu_Gk)[108] = 0.0;
  (*gpu_Gk)[124] = T;
  (*gpu_Gk)[13] = T;
  (*gpu_Gk)[29] = 0.0;
  (*gpu_Gk)[45] = T;
  (*gpu_Gk)[61] = 0.0;
  (*gpu_Gk)[77] = 0.0;
  (*gpu_Gk)[93] = T;
  (*gpu_Gk)[109] = 0.0;
  (*gpu_Gk)[125] = T;
  (*gpu_Gk)[14] = 0.0;
  (*gpu_Gk)[30] = T * T / 2.0;
  (*gpu_Gk)[46] = 0.0;
  (*gpu_Gk)[62] = T * T / 2.0;
  (*gpu_Gk)[78] = 0.0;
  (*gpu_Gk)[94] = T;
  (*gpu_Gk)[110] = 0.0;
  (*gpu_Gk)[126] = T;
  (*gpu_Gk)[15] = 0.0;
  (*gpu_Gk)[31] = T;
  (*gpu_Gk)[47] = 0.0;
  (*gpu_Gk)[63] = T;
  (*gpu_Gk)[79] = 0.0;
  (*gpu_Gk)[95] = T;
  (*gpu_Gk)[111] = 0.0;
  (*gpu_Gk)[127] = T;

  //  Prediction step alternative 2. Gives easier matrix to invert
  e_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_Fk_inv, *gpu_Pk_init);
  syncIsDirty = true;
  for (c = 0; c < 16; c++) {
    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_C)[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 16; c++) {
    jp1j = c * 16 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_C)[k] += (*b_gpu_Ik)[b_c - 1] * (*gpu_Pk_init)[ia];
      }

      jy += 16;
    }

    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_Ih)[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 16; c++) {
    jp1j = c * 16 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_Ih)[k] += (*gpu_Fk_inv)[b_c - 1] * (*gpu_C)[ia];
      }

      jy += 16;
    }
  }

  // 4*4
  for (c = 0; c < 8; c++) {
    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      C[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 8; c++) {
    jp1j = c * 16 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        C[k] += (*gpu_Gk)[b_c - 1] * (*gpu_Ih)[ia];
      }

      jy += 16;
    }
  }

  f_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(128U, 1U, 1U)>>>
    (*gpu_Gk, *gpu_A);
  syncIsDirty = true;
  for (c = 0; c < 16; c++) {
    iy = c << 3;
    i1 = iy - 7;
    for (ic = 0; ic <= iy - i1; ic++) {
      b_C[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 16; c++) {
    jp1j = c * 16 + 1;
    iy = c << 3;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 6;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        b_C[k] += (*gpu_Ih)[b_c - 1] * (*gpu_A)[ia];
      }

      jy += 8;
    }
  }

  for (c = 0; c < 8; c++) {
    iy = c << 3;
    i1 = iy - 7;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_Rw)[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 8; c++) {
    jp1j = c * 16 + 1;
    iy = c << 3;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 6;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_Rw)[k] += (*gpu_Gk)[b_c - 1] * b_C[ia];
      }

      jy += 8;
    }
  }

  g_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Rw_inv, *gpu_Rw, *gpu_y);
  h_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 7; j++) {
    c = j * 9;
    jp1j = c - 4;
    b_c = 6 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Rw)[c]);
    for (k = 0; k <= b_c; k++) {
      ix++;
      s = std::abs((*gpu_Rw)[ix]);
      if (s > smax) {
        iy = k + 1;
        smax = s;
      }
    }

    if ((*gpu_Rw)[c + iy] != 0.0) {
      if (iy != 0) {
        (*b_gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        b_c = iy;
        for (k = 0; k < 8; k++) {
          ix = j + k * 8;
          iy = b_c + k * 8;
          smax = (*gpu_Rw)[ix];
          (*gpu_Rw)[ix] = (*gpu_Rw)[iy];
          (*gpu_Rw)[iy] = smax;
        }
      }

      i1 = (c - j) + 2;
      for (i = 0; i <= i1 - jp1j; i++) {
        iy = (c + i) + 1;
        (*gpu_Rw)[iy] /= (*gpu_Rw)[c];
      }
    }

    b_c = 6 - j;
    iy = c + 10;
    jy = c + 8;
    for (ia = 0; ia <= b_c; ia++) {
      smax = (*gpu_Rw)[jy];
      if ((*gpu_Rw)[jy] != 0.0) {
        ix = c;
        i1 = iy - 6;
        i2 = iy - j;
        for (jp1j = 0; jp1j <= i2 - i1; jp1j++) {
          i = (iy + jp1j) - 1;
          (*gpu_Rw)[i] += (*gpu_Rw)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 8;
      iy += 8;
    }
  }

  i_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 7; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*b_gpu_ipiv)[k] > k + 1) {
      iy = (*gpu_p)[(*b_gpu_ipiv)[k] - 1];
      (*gpu_p)[(*b_gpu_ipiv)[k] - 1] = (*gpu_p)[k];
      (*gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 8; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    b_i = (*gpu_p)[k];
    (*gpu_y)[k + (((*gpu_p)[k] - 1) << 3)] = 1.0;
    for (j = 0; j <= 7 - k; j++) {
      ia = k + j;
      if ((*gpu_y)[ia + ((b_i - 1) << 3)] != 0.0) {
        for (i = 0; i <= 6 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_y)[iy + ((b_i - 1) << 3)] -= (*gpu_y)[ia + ((b_i - 1) << 3)] * (*
            gpu_Rw)[iy + (ia << 3)];
        }
      }
    }
  }

  for (j = 0; j < 8; j++) {
    jp1j = (j << 3) - 1;
    for (k = 0; k < 8; k++) {
      iy = 8 - k;
      jy = (7 - k) << 3;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_y)[(jp1j - k) + 8] != 0.0) {
        (*gpu_y)[(jp1j - k) + 8] /= (*gpu_Rw)[(jy - k) + 7];
        for (i = 0; i <= iy - 2; i++) {
          (*gpu_y)[(i + jp1j) + 1] -= (*gpu_y)[(jp1j - k) + 8] * (*gpu_Rw)[i +
            jy];
        }
      }
    }

    c = j << 4;
    i1 = c - 15;
    for (ic = 0; ic <= c - i1; ic++) {
      c_C[c + ic] = 0.0;
    }
  }

  for (c = 0; c < 8; c++) {
    jp1j = c * 8 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 7;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        c_C[k] += (*gpu_y)[b_c - 1] * C[ia];
      }

      jy += 16;
    }
  }

  j_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(128U, 1U, 1U)>>>
    (*gpu_Gk, *gpu_A);
  syncIsDirty = true;
  for (c = 0; c < 16; c++) {
    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_Pk_init)[iy + ic] = 0.0;
    }
  }

  // 4*4
  for (c = 0; c < 16; c++) {
    jp1j = c * 8 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 7;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_Pk_init)[k] += (*gpu_A)[b_c - 1] * c_C[ia];
      }

      jy += 16;
    }

    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_C)[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 16; c++) {
    jp1j = c * 16 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 15;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_C)[k] += (*gpu_Ih)[b_c - 1] * (*gpu_Pk_init)[ia];
      }

      jy += 16;
    }
  }

  // 4*4
  if (syncIsDirty) {
    hipDeviceSynchronize();
  }

  (*gpu_ih)[0] = initialization_vec[2];
  (*gpu_ih)[1] = 0.0;
  (*gpu_ih)[2] = 0.0;
  (*gpu_ih)[3] = 0.0;
  (*gpu_ih)[4] = initialization_vec[2];
  (*gpu_ih)[5] = 0.0;
  (*gpu_ih)[6] = 0.0;
  (*gpu_ih)[7] = 0.0;
  (*gpu_ih)[8] = initialization_vec[2];
  (*gpu_ih)[9] = 0.0;
  (*gpu_ih)[10] = 0.0;
  (*gpu_ih)[11] = 0.0;
  (*gpu_ih)[12] = initialization_vec[2];
  k_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih);
  l_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih, *b_gpu_Ik, *c_gpu_Ik);
  m_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*c_gpu_Ik, *gpu_Fk_inv, *gpu_ih);

  // 4*1
  n_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Pk_init, *gpu_ih, *gpu_op);

  // 4*1
  o_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (*gpu_C, *gpu_Ih, *gpu_Pk_init);
  p_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 15; j++) {
    c = j * 17;
    jp1j = c - 12;
    b_c = 14 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Ih)[c]);
    for (k = 0; k <= b_c; k++) {
      ix++;
      s = std::abs((*gpu_Ih)[ix]);
      if (s > smax) {
        iy = k + 1;
        smax = s;
      }
    }

    if ((*gpu_Ih)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        b_c = iy;
        for (k = 0; k < 16; k++) {
          ix = j + k * 16;
          iy = b_c + k * 16;
          smax = (*gpu_Ih)[ix];
          (*gpu_Ih)[ix] = (*gpu_Ih)[iy];
          (*gpu_Ih)[iy] = smax;
        }
      }

      i1 = (c - j) + 2;
      for (i = 0; i <= i1 - jp1j; i++) {
        iy = (c + i) + 1;
        (*gpu_Ih)[iy] /= (*gpu_Ih)[c];
      }
    }

    b_c = 14 - j;
    iy = c + 18;
    jy = c + 16;
    for (ia = 0; ia <= b_c; ia++) {
      smax = (*gpu_Ih)[jy];
      if ((*gpu_Ih)[jy] != 0.0) {
        ix = c;
        i1 = iy - 14;
        i2 = iy - j;
        for (jp1j = 0; jp1j <= i2 - i1; jp1j++) {
          i = (iy + jp1j) - 1;
          (*gpu_Ih)[i] += (*gpu_Ih)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 16;
      iy += 16;
    }
  }

  q_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_p);
  syncIsDirty = true;
  for (k = 0; k < 15; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[k] > k + 1) {
      iy = (*b_gpu_p)[(*gpu_ipiv)[k] - 1];
      (*b_gpu_p)[(*gpu_ipiv)[k] - 1] = (*b_gpu_p)[k];
      (*b_gpu_p)[k] = static_cast<signed char>(iy);
    }
  }

  for (k = 0; k < 16; k++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    b_i = (*b_gpu_p)[k];
    (*gpu_Pk_init)[k + (((*b_gpu_p)[k] - 1) << 4)] = 1.0;
    for (j = 0; j <= 15 - k; j++) {
      ia = k + j;
      if ((*gpu_Pk_init)[ia + ((b_i - 1) << 4)] != 0.0) {
        for (i = 0; i <= 14 - ia; i++) {
          iy = (ia + i) + 1;
          (*gpu_Pk_init)[iy + ((b_i - 1) << 4)] -= (*gpu_Pk_init)[ia + ((b_i - 1)
            << 4)] * (*gpu_Ih)[iy + (ia << 4)];
        }
      }
    }
  }

  for (j = 0; j < 16; j++) {
    jp1j = (j << 4) - 1;
    for (k = 0; k < 16; k++) {
      iy = 16 - k;
      jy = (15 - k) << 4;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Pk_init)[(jp1j - k) + 16] != 0.0) {
        (*gpu_Pk_init)[(jp1j - k) + 16] /= (*gpu_Ih)[(jy - k) + 15];
        for (i = 0; i <= iy - 2; i++) {
          (*gpu_Pk_init)[(i + jp1j) + 1] -= (*gpu_Pk_init)[(jp1j - k) + 16] *
            (*gpu_Ih)[i + jy];
        }
      }
    }
  }

  // 4*4
  r_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_op, *gpu_Pk_init, *gpu_xk_m_out);

  // 4*1
  // ---- Measurement step --------------------------
  //  Extract front and rear track widths
  //  drive axle 1
  //  drive axle 2
  smax = L_axlePos[0] + L_geometricWheelbase;

  // 1*1
  s_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_B_usedMeas_vec, *gpu_Re, *gpu_Re_inv);

  // 1*13
  t_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>
    (*b_gpu_Re_inv);
  u_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Re_inv, *b_gpu_Re_inv);

  // 13*13
  //  Linear parts
  hipDeviceSynchronize();
  (*gpu_H_linear)[1] = 0.0;
  (*gpu_H_linear)[14] = 0.0;
  (*gpu_H_linear)[27] = 0.0;
  (*gpu_H_linear)[40] = L_imuToRear;
  (*gpu_H_linear)[53] = 0.0;
  (*gpu_H_linear)[66] = 0.0;
  (*gpu_H_linear)[79] = 0.0;
  (*gpu_H_linear)[92] = L_imuToRear;
  (*gpu_H_linear)[105] = 0.0;
  (*gpu_H_linear)[118] = 0.0;
  (*gpu_H_linear)[131] = 0.0;
  (*gpu_H_linear)[144] = L_imuToRear;
  (*gpu_H_linear)[157] = 0.0;
  (*gpu_H_linear)[170] = 0.0;
  (*gpu_H_linear)[183] = 0.0;
  (*gpu_H_linear)[196] = L_imuToRear;
  (*gpu_H_linear)[6] = 1.0;
  (*gpu_H_linear)[19] = 0.0;
  (*gpu_H_linear)[32] = -L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[45] = 0.0;
  (*gpu_H_linear)[58] = 1.0;
  (*gpu_H_linear)[71] = 0.0;
  (*gpu_H_linear)[84] = -L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[97] = 0.0;
  (*gpu_H_linear)[110] = 1.0;
  (*gpu_H_linear)[123] = 0.0;
  (*gpu_H_linear)[136] = 0.0;
  (*gpu_H_linear)[149] = 0.0;
  (*gpu_H_linear)[162] = 1.0;
  (*gpu_H_linear)[175] = 0.0;
  (*gpu_H_linear)[188] = 0.0;
  (*gpu_H_linear)[201] = 0.0;
  (*gpu_H_linear)[7] = 1.0;
  (*gpu_H_linear)[20] = 0.0;
  (*gpu_H_linear)[33] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[46] = 0.0;
  (*gpu_H_linear)[59] = 1.0;
  (*gpu_H_linear)[72] = 0.0;
  (*gpu_H_linear)[85] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[98] = 0.0;
  (*gpu_H_linear)[111] = 1.0;
  (*gpu_H_linear)[124] = 0.0;
  (*gpu_H_linear)[137] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[150] = 0.0;
  (*gpu_H_linear)[163] = 1.0;
  (*gpu_H_linear)[176] = 0.0;
  (*gpu_H_linear)[189] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[202] = 0.0;
  (*gpu_H_linear)[8] = 1.0;
  (*gpu_H_linear)[21] = 0.0;
  (*gpu_H_linear)[34] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[47] = 0.0;
  (*gpu_H_linear)[60] = 1.0;
  (*gpu_H_linear)[73] = 0.0;
  (*gpu_H_linear)[86] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[99] = 0.0;
  (*gpu_H_linear)[112] = 1.0;
  (*gpu_H_linear)[125] = 0.0;
  (*gpu_H_linear)[138] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[151] = 0.0;
  (*gpu_H_linear)[164] = 1.0;
  (*gpu_H_linear)[177] = 0.0;
  (*gpu_H_linear)[190] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[203] = 0.0;
  (*gpu_H_linear)[9] = 1.0;
  (*gpu_H_linear)[22] = 0.0;
  (*gpu_H_linear)[35] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[48] = 0.0;
  (*gpu_H_linear)[61] = 1.0;
  (*gpu_H_linear)[74] = 0.0;
  (*gpu_H_linear)[87] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[100] = 0.0;
  (*gpu_H_linear)[113] = 1.0;
  (*gpu_H_linear)[126] = 0.0;
  (*gpu_H_linear)[139] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[152] = 0.0;
  (*gpu_H_linear)[165] = 1.0;
  (*gpu_H_linear)[178] = 0.0;
  (*gpu_H_linear)[191] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[204] = 0.0;
  (*gpu_H_linear)[10] = 1.0;
  (*gpu_H_linear)[23] = 0.0;
  (*gpu_H_linear)[36] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[49] = 0.0;
  (*gpu_H_linear)[62] = 1.0;
  (*gpu_H_linear)[75] = 0.0;
  (*gpu_H_linear)[88] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[101] = 0.0;
  (*gpu_H_linear)[114] = 1.0;
  (*gpu_H_linear)[127] = 0.0;
  (*gpu_H_linear)[140] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[153] = 0.0;
  (*gpu_H_linear)[166] = 1.0;
  (*gpu_H_linear)[179] = 0.0;
  (*gpu_H_linear)[192] = L_trackWidth[3] / 2.0;
  v_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_H_linear);
  w_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv8, *gpu_iv9, *gpu_iv10, *gpu_iv11, *gpu_H_linear);

  //  Nonlinear parts
  x_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_xk_m_out, *gpu_H_linear, *gpu_hk);

  // 13*1
  // the new hk 13*1 matrix is formed from this calculation
  hipDeviceSynchronize();
  (*gpu_hk)[4] = ((*gpu_xk_m_out)[0] - L_trackWidth[0] * (*gpu_xk_m_out)[2] /
                  2.0) * std::cos(delta) + smax * (*gpu_xk_m_out)[2] * std::sin
    (delta);
  (*gpu_hk)[5] = ((*gpu_xk_m_out)[0] + L_trackWidth[0] * (*gpu_xk_m_out)[2] /
                  2.0) * std::cos(delta) + smax * (*gpu_xk_m_out)[2] * std::sin
    (delta);
  (*gpu_H_linear)[30] = -L_trackWidth[0] * std::cos(delta) / 2.0 + smax * std::
    sin(delta);
  (*gpu_H_linear)[5] = std::cos(delta);
  (*gpu_H_linear)[31] = L_trackWidth[0] * std::cos(delta) / 2.0 + smax * std::
    sin(delta);
  y_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(smax,
    delta, L_imuToRear, *gpu_xk_m_out, *gpu_H_linear, *gpu_hk);
  ab_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
    (*gpu_H_linear, *b_gpu_A);
  syncIsDirty = true;
  for (c = 0; c < 13; c++) {
    iy = c << 4;
    i1 = iy - 15;
    for (ic = 0; ic <= iy - i1; ic++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*b_gpu_C)[iy + ic] = 0.0;
    }
  }

  for (c = 0; c < 13; c++) {
    jp1j = c * 13 + 1;
    iy = c << 4;
    jy = -1;
    i1 = jp1j - 12;
    for (i = 0; i <= jp1j - i1; i++) {
      b_c = jp1j + i;
      ia = jy;
      i2 = iy - 14;
      ix = iy + 1;
      for (ic = 0; ic <= ix - i2; ic++) {
        k = iy + ic;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*b_gpu_C)[k] += (*b_gpu_Re_inv)[b_c - 1] * (*b_gpu_A)[ia];
      }

      jy += 16;
    }
  }

  bb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_xk_m_out, *gpu_H_linear, *gpu_hk, *gpu_y_meas, *b_gpu_y_meas);
  cb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_y_meas, *b_gpu_C, *gpu_op);

  // Hk is always a 13*4 matrix and hk is always a 13*1 matrix
  hipDeviceSynchronize();
  hipMemcpy(&op[0], gpu_op, 128ULL, hipMemcpyDeviceToHost);
  hipMemcpy(&xk_m_out[0], gpu_xk_m_out, 128ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_y_meas);
  hipFree(*gpu_iv11);
  hipFree(*gpu_iv10);
  hipFree(*gpu_iv9);
  hipFree(*gpu_iv8);
  hipFree(*gpu_Re);
  hipFree(*gpu_B_usedMeas_vec);
  hipFree(*gpu_iv7);
  hipFree(*gpu_iv6);
  hipFree(*gpu_iv5);
  hipFree(*gpu_iv4);
  hipFree(*gpu_iv3);
  hipFree(*gpu_iv2);
  hipFree(*gpu_iv1);
  hipFree(*gpu_iv);
  hipFree(*b_gpu_Rw);
  hipFree(*gpu_Ik);
  hipFree(*gpu_op);
  hipFree(*gpu_xk_m_out);
  hipFree(*gpu_Pk_init);
  hipFree(*gpu_ipiv);
  hipFree(*b_gpu_Ik);
  hipFree(*gpu_Rw);
  hipFree(*gpu_Rw_inv);
  hipFree(*b_gpu_ipiv);
  hipFree(*gpu_p);
  hipFree(*gpu_Fk_inv);
  hipFree(*b_gpu_p);
  hipFree(*gpu_Gk);
  hipFree(*gpu_A);
  hipFree(*gpu_y);
  hipFree(*gpu_ih);
  hipFree(*c_gpu_Ik);
  hipFree(*gpu_C);
  hipFree(*gpu_Ih);
  hipFree(*gpu_Re_inv);
  hipFree(*b_gpu_Re_inv);
  hipFree(*gpu_H_linear);
  hipFree(*gpu_hk);
  hipFree(*b_gpu_A);
  hipFree(*b_gpu_y_meas);
  hipFree(*b_gpu_C);
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_initialize()
{
  state_not_empty = false;
  eml_rand_init();
  eml_rand_mcg16807_stateful_init();
  eml_rand_shr3cong_stateful_init();
  isInitialized_InformationFilterUpdate = true;
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_terminate()
{
  // (no terminate code required)
  isInitialized_InformationFilterUpdate = false;
}

//
// File trailer for InformationFilterUpdate.cu
//
// [EOF]
//
